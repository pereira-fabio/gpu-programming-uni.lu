// Copyright 2023 Pierre Talbot


#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void hello_world() {
  printf("hello world!\n");
}

int main(int argc, char** argv) {
  hello_world<<<1, 1>>>();
  hipDeviceSynchronize();
  return 0;
}
