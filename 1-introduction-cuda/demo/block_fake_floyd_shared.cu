#include "hip/hip_runtime.h"
// Copyright 2023 Pierre Talbot

#include "../utility.hpp"
#include <string>

__forceinline__ __device__ int dim2D(int x, int y, int n) {
  return x * n + y;
}

__global__ void floyd_warshall_gpu(int** d, size_t n) {
  for(int kk = 0; kk < n*n; ++kk) { // just to show that shared memory can help... Not useful.
    for(int k = 0; k < n; ++k) {
      for(int i = 0; i < n; ++i) {
        for(int j = threadIdx.x; j < n; j += blockDim.x) {
          if(d[i][j] > d[i][k] + d[k][j]) {
            d[i][j] = d[i][k] + d[k][j];
          }
        }
      }
      __syncthreads();
    }
  }
}

__global__ void floyd_warshall_gpu_shared(int** d, size_t n) {
  // Copy the matrix into the shared memory.
  extern __shared__ int d2[];
  for(int i = 0; i < n; ++i) {
    for(int j = threadIdx.x; j < n; j += blockDim.x) {
      d2[dim2D(i, j, n)] = d[i][j];
    }
  }
  __syncthreads();
  // Compute on the shared memory.
  for(int kk = 0; kk < n*n; ++kk) { // just to show that shared memory can help... Not useful.
    for(int k = 0; k < n; ++k) {
      for(int i = 0; i < n; ++i) {
        for(int j = threadIdx.x; j < n; j += blockDim.x) {
          if(d2[dim2D(i,j,n)] > d2[dim2D(i,k,n)] + d2[dim2D(k,j,n)]) {
            d2[dim2D(i,j,n)] = d2[dim2D(i,k,n)] + d2[dim2D(k,j,n)];
          }
        }
      }
      __syncthreads();
    }
  }
  // Copy the matrix back to the global memory.
  for(int i = 0; i < n; ++i) {
    for(int j = threadIdx.x; j < n; j += blockDim.x) {
      d[i][j] = d2[dim2D(i, j, n)];
    }
  }
}

template <class T>
void floyd_warshall_cpu(std::vector<std::vector<T>>& d) {
  size_t n = d.size();
  for(int kk = 0; kk < n*n; ++kk) { // just to show that shared memory can help... Not useful.
    for(int k = 0; k < n; ++k) {
      for(int i = 0; i < n; ++i) {
        for(int j = 0; j < n; ++j) {
          if(d[i][j] > d[i][k] + d[k][j]) {
            d[i][j] = d[i][k] + d[k][j];
          }
        }
      }
    }
  }
}

int main(int argc, char** argv) {
  if(argc != 3) {
    std::cout << "usage: " << argv[0] << " <matrix size> <threads-per-block>" << std::endl;
    exit(1);
  }
  size_t n = std::stoi(argv[1]);
  size_t threads_per_block = std::stoi(argv[2]);

  // I. Generate a random distance matrix of size N x N.
  std::vector<std::vector<int>> cpu_distances = initialize_distances(n);
  // Note that `std::vector` cannot be used on GPU, hence we transfer it into a simple `int**` array in managed memory.
  int** gpu_distances1 = initialize_gpu_distances(cpu_distances);
  int** gpu_distances2 = initialize_gpu_distances(cpu_distances);

  // II. Running Floyd Warshall on CPU.
  long cpu_ms = benchmark_one_ms([&]{
    floyd_warshall_cpu(cpu_distances);
  });
  std::cout << "CPU: " << cpu_ms << " ms" << std::endl;

  // III. Running Floyd Warshall on GPU (single block of size `threads_per_block`).

  /** Maximal capacity of the shared memory. */
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  size_t shared_mem_capacity = deviceProp.sharedMemPerBlock;
  size_t matrix_size = n * n * sizeof(int);
  if(shared_mem_capacity < matrix_size) {
    std::cerr << "matrix too large to be in shared memory." << std::endl;
    exit(1);
  }

  long gpu_ms = benchmark_one_ms([&]{
    floyd_warshall_gpu<<<1, threads_per_block>>>(gpu_distances1, n);
    CUDIE(hipDeviceSynchronize());
  });
  std::cout << "GPU: " << gpu_ms << " ms" << std::endl;

  long gpu_shared_ms = benchmark_one_ms([&]{
    floyd_warshall_gpu_shared<<<1, threads_per_block, matrix_size>>>(gpu_distances2, n);
    CUDIE(hipDeviceSynchronize());
  });
  std::cout << "GPU: " << gpu_shared_ms << " ms" << std::endl;

  // IV. Verifying both give the same result and deallocating.
  check_equal_matrix(cpu_distances, gpu_distances2);
  deallocate_gpu_distances(gpu_distances1, n);
  deallocate_gpu_distances(gpu_distances2, n);
  return 0;
}
