#include "hip/hip_runtime.h"
// Copyright 2023 Pierre Talbot

#include "../utility.hpp"
#include <string>

__global__ void floyd_warshall_gpu(int** d, size_t n, size_t k) {
  for(int i = blockIdx.x; i < n; i += gridDim.x) {
    for(int j = threadIdx.x; j < n; j += blockDim.x) {
      if(d[i][j] > d[i][k] + d[k][j]) {
        d[i][j] = d[i][k] + d[k][j];
      }
    }
  }
}

void floyd_warshall_cpu(std::vector<std::vector<int>>& d) {
  size_t n = d.size();
  for(int k = 0; k < n; ++k) {
    for(int i = 0; i < n; ++i) {
      for(int j = 0; j < n; ++j) {
        if(d[i][j] > d[i][k] + d[k][j]) {
          d[i][j] = d[i][k] + d[k][j];
        }
      }
    }
  }
}

int main(int argc, char** argv) {
  if(argc != 4) {
    std::cout << "usage: " << argv[0] << " <matrix size> <threads-per-block> <num-blocks>" << std::endl;
    exit(1);
  }
  size_t n = std::stoi(argv[1]);
  size_t threads_per_block = std::stoi(argv[2]);
  size_t num_blocks = std::stoi(argv[3]);

  // I. Generate a random distance matrix of size N x N.
  std::vector<std::vector<int>> cpu_distances = initialize_distances(n);
  // Note that `std::vector` cannot be used on GPU, hence we transfer it into a simple `int**` array in managed memory.
  int** gpu_distances = initialize_gpu_distances(cpu_distances);

  // II. Running Floyd Warshall on CPU.
  long cpu_ms = benchmark_one_ms([&]{
    floyd_warshall_cpu(cpu_distances);
  });
  std::cout << "CPU: " << cpu_ms << " ms" << std::endl;

  // III. Running Floyd Warshall on the whole GPU grid.
  long gpu_ms = benchmark_one_ms([&]{
    for(int k = 0; k < n; ++k) {
      floyd_warshall_gpu<<<num_blocks, threads_per_block>>>(gpu_distances, n, k);
      CUDIE(hipDeviceSynchronize());
    }
  });
  std::cout << "GPU: " << gpu_ms << " ms" << std::endl;

  // IV. Verifying both give the same result and deallocating.
  check_equal_matrix(cpu_distances, gpu_distances);
  deallocate_gpu_distances(gpu_distances, n);
  return 0;
}
